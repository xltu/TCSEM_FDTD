#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               		TCSEM_FDTD	version  2.0
 !               ---------------------------------------
 ! This is the CUDA version of the TCSEM_FDTD program
 !
 ! TCSEM_FDTD is a time domain modeling code for marine controlled source
 ! electromagnetic method. The code models the impulse response generated 
 ! by a electrical bipole souce towed in the seawater. All six components 
 ! (i.e., Ex, Ey, Ez, Bx, By, and Bz) of the EM field could be obtained 
 ! simultaneously from one forward modeling.
 !
 ! The FDTD modeling uses dufort frankel scheme, and staggered grid
 ! The source was added in a similar way as in seismic modeling
 ! The waveform of a delt function is approximated by a Gaussian function
 ! The output of the programs is the impulse response
 !
 ! TODO In the current version, multiple transmitter positions are Looped 
 ! on one GPU device squentially. This should be changed to parallel
 ! threads on multiple GPU devices in feature version
 !
 ! XXX Check README to prepare the input files required by this program
 !
 ! Created Dec.28.2020 by Xiaolei Tu
 ! Send bug reports, comments or suggestions to tuxl2009@hotmail.com
 !=====================================================================
 */

/* The current header file is used to deal with the GPU device and
	CUDA version related issues.
   It is modified from 'prepare_mesh_constants_cuda.h' from the program
   'specfem3d version 3.0' at https://github.com/geodynamics/specfem3d
*/

#include "Cuda_device.h"
#include"FTDT.h"
#include<math.h>
/* ----------------------------------------------------------------------------------------------- */

// GPU preparation

/* ----------------------------------------------------------------------------------------------- */
  
void prepare_device_arrays(DArrays* DPtrs, ModelPara MP, realw ***HostCond, 
													 RxPos *d_xyzRx, RxPos xyzRx, int NRx, int **RxLst,
													 ModelGrid *d_MG, ModelGrid MG, GridConv *d_GC, GridConv GC)
{

  TRACE("prepare device arrays");
	int L,M,N,L1,M1;
	
	L=MP.L;
	M=MP.M;
	N=MP.N;
	
	L1=MP.AnLx_New+MP.AnRx_New+MP.Nx;
  M1=2*MP.Any_New+MP.Ny;
#if MAXDEBUG == 1
	printf("L=%d, M=%d, N=%d\n",L,M,N);
#endif				
	// EM field
	print_CUDA_error_if_any( hipMalloc((void **)&(DPtrs->Ex), L*(M+1)*(N+1)*sizeof(realw)), 1001);
	print_CUDA_error_if_any( hipMalloc((void **)&(DPtrs->Ey), (L+1)*(M)*(N+1)*sizeof(realw)), 1002);
	print_CUDA_error_if_any( hipMalloc((void **)&(DPtrs->Ez), (L+1)*(M+1)*(N)*sizeof(realw)), 1003);
	print_CUDA_error_if_any( hipMalloc((void **)&(DPtrs->Bx), (L+1)*(M)*(N)*sizeof(realw)), 1004);
	print_CUDA_error_if_any( hipMalloc((void **)&(DPtrs->By), L*(M+1)*(N)*sizeof(realw)), 1005);
	print_CUDA_error_if_any( hipMalloc((void **)&(DPtrs->Bz), L*(M)*(N+1)*sizeof(realw)), 1006);
	
	// Mem for Bx air and By air, 2D padded Mem
	// XXX may consider using texture mem in a future version

	print_CUDA_error_if_any( hipMallocPitch(&(DPtrs->BxAir), &(DPtrs->BxPitch), (L+1)*sizeof(realw), M), 1007);
	print_CUDA_error_if_any( hipMallocPitch(&(DPtrs->ByAir), &(DPtrs->ByPitch), (L)*sizeof(realw), M+1), 1008);	
	
	TRACE("Copy conductivity model from host to device\n");
	//conductivity
	print_CUDA_error_if_any( hipMalloc((void **)&(DPtrs->Con), L*(M)*(N)*sizeof(realw)), 1011);
	print_CUDA_error_if_any( hipMemcpy(DPtrs->Con, HostCond[0][0], L*(M)*(N)*sizeof(realw), hipMemcpyHostToDevice), 1012 );
	// free host conductivity 
	Free_3D_Array(HostCond,N);
	
	//Structure of arrays
	// https://stackoverflow.com/questions/31598021/cuda-cudamemcpy-struct-of-arrays
	TRACE("Copy receiver arrays from host to device\n");
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_xyzRx->x), NRx*sizeof(realw)), 1013);
	print_CUDA_error_if_any( hipMemcpy(d_xyzRx->x, xyzRx.x, NRx*sizeof(realw), hipMemcpyHostToDevice), 1014 );
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_xyzRx->y), NRx*sizeof(realw)), 1015);
	print_CUDA_error_if_any( hipMemcpy(d_xyzRx->y, xyzRx.y, NRx*sizeof(realw), hipMemcpyHostToDevice), 1016 );
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_xyzRx->z), NRx*sizeof(realw)), 1017);
	print_CUDA_error_if_any( hipMemcpy(d_xyzRx->z, xyzRx.z, NRx*sizeof(realw), hipMemcpyHostToDevice), 1018 );
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_xyzRx->ix), NRx*sizeof(int)), 1019);
	print_CUDA_error_if_any( hipMemcpy(d_xyzRx->ix, xyzRx.ix, NRx*sizeof(int), hipMemcpyHostToDevice), 1020 );
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_xyzRx->iy), NRx*sizeof(int)), 1021);
	print_CUDA_error_if_any( hipMemcpy(d_xyzRx->iy, xyzRx.iy, NRx*sizeof(int), hipMemcpyHostToDevice), 1022 );
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_xyzRx->iz), NRx*sizeof(int)), 1023);
	print_CUDA_error_if_any( hipMemcpy(d_xyzRx->iz, xyzRx.iz, NRx*sizeof(int), hipMemcpyHostToDevice), 1024 );
	
	
	// allocate RxLst on device
	print_CUDA_error_if_any( hipMalloc((void **)RxLst, NRx*sizeof(int)), 1025);
	
	TRACE("Allocate model grid arrays on device\n");
	// allocate Model Grid arrays
	print_CUDA_error_if_any( hipMalloc((void **)&(d_MG->dx), L*sizeof(realw)), 1027);
	print_CUDA_error_if_any( hipMemcpy(d_MG->dx, MG.dx, L*sizeof(realw), hipMemcpyHostToDevice), 1028 );
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_MG->dy), M*sizeof(realw)), 1029);
	print_CUDA_error_if_any( hipMemcpy(d_MG->dy, MG.dy, M*sizeof(realw), hipMemcpyHostToDevice), 1030 );
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_MG->dz), N*sizeof(realw)), 1031);
	print_CUDA_error_if_any( hipMemcpy(d_MG->dz, MG.dz, N*sizeof(realw), hipMemcpyHostToDevice), 1032 );
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_MG->X_Bzold), L*sizeof(realw)), 1033);
	print_CUDA_error_if_any( hipMemcpy(d_MG->X_Bzold, MG.X_Bzold, L*sizeof(realw), hipMemcpyHostToDevice), 1034 );
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_MG->Y_Bzold), M*sizeof(realw)), 1035);
	print_CUDA_error_if_any( hipMemcpy(d_MG->Y_Bzold, MG.Y_Bzold, M*sizeof(realw), hipMemcpyHostToDevice), 1036 );
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_MG->Z), N*sizeof(realw)), 1037);
	print_CUDA_error_if_any( hipMemcpy(d_MG->Z, MG.Z, N*sizeof(realw), hipMemcpyHostToDevice), 1038 );	
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_MG->X_BzNew), (MP.AnLx_New+MP.AnRx_New+MP.Nx)*sizeof(realw)), 1039);
	print_CUDA_error_if_any( hipMemcpy(d_MG->X_BzNew, MG.X_BzNew, (MP.AnLx_New+MP.AnRx_New+MP.Nx)*sizeof(realw), hipMemcpyHostToDevice), 1040 );
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_MG->Y_BzNew), (2*MP.Any_New+MP.Ny)*sizeof(realw)), 1041);
	print_CUDA_error_if_any( hipMemcpy(d_MG->Y_BzNew, MG.Y_BzNew, (2*MP.Any_New+MP.Ny)*sizeof(realw), hipMemcpyHostToDevice), 1042 );
	
	
	//alocate grid mapping arrays
	print_CUDA_error_if_any( hipMalloc((void **)&(d_GC->xBzNew2old), L1*sizeof(int)), 1201);
	print_CUDA_error_if_any( hipMemcpy(d_GC->xBzNew2old, GC.xBzNew2old, L1*sizeof(int), hipMemcpyHostToDevice), 1202 );
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_GC->yBzNew2old), M1*sizeof(int)), 1203);
	print_CUDA_error_if_any( hipMemcpy(d_GC->yBzNew2old, GC.yBzNew2old, M1*sizeof(int), hipMemcpyHostToDevice), 1204 );
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_GC->xBzold2New), L*sizeof(int)), 1205);
	print_CUDA_error_if_any( hipMemcpy(d_GC->xBzold2New, GC.xBzold2New, L*sizeof(int), hipMemcpyHostToDevice), 1206 );
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_GC->yBzold2New), M*sizeof(int)), 1207);
	print_CUDA_error_if_any( hipMemcpy(d_GC->yBzold2New, GC.yBzold2New, M*sizeof(int), hipMemcpyHostToDevice), 1208 );
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_GC->xoldBx2NewBz), (L+1)*sizeof(int)), 1209);
	print_CUDA_error_if_any( hipMemcpy(d_GC->xoldBx2NewBz, GC.xoldBx2NewBz, (L+1)*sizeof(int), hipMemcpyHostToDevice), 1210 );
	
	print_CUDA_error_if_any( hipMalloc((void **)&(d_GC->yoldBy2NewBz), (M+1)*sizeof(int)), 1211);
	print_CUDA_error_if_any( hipMemcpy(d_GC->yoldBy2NewBz, GC.yoldBy2NewBz, (M+1)*sizeof(int), hipMemcpyHostToDevice), 1212 );
	
  GPU_ERROR_CHECKING("prepare device arrays");
}


/* ----------------------------------------------------------------------------------------------- */

// set device memory to zeros

/* ----------------------------------------------------------------------------------------------- */
void set_zeros_EM_arrays(DArrays* DPtrs, ModelPara *MP)
{

  TRACE("set EM field (device arrays) to zeros");
	int L,M,N;
	
	L=MP->L;
	M=MP->M;
	N=MP->N;
				
	// EM field
	print_CUDA_error_if_any( hipMemset( DPtrs->Ex, 0, L*(M+1)*(N+1)*sizeof(realw)), 1101);
	print_CUDA_error_if_any( hipMemset( DPtrs->Ey, 0, (L+1)*(M)*(N+1)*sizeof(realw)), 1102);
	print_CUDA_error_if_any( hipMemset( DPtrs->Ez, 0, (L+1)*(M+1)*(N)*sizeof(realw)), 1103);
	print_CUDA_error_if_any( hipMemset( DPtrs->Bx, 0, (L+1)*(M)*(N)*sizeof(realw)), 1104);
	print_CUDA_error_if_any( hipMemset( DPtrs->By, 0, (L)*(M+1)*(N)*sizeof(realw)), 1105);
	print_CUDA_error_if_any( hipMemset( DPtrs->Bz, 0, (L)*(M)*(N+1)*sizeof(realw)), 1106);
		
	print_CUDA_error_if_any( hipMemset2D((DPtrs->BxAir), (DPtrs->BxPitch), 0, (L+1)*sizeof(realw), M), 1107);
	print_CUDA_error_if_any( hipMemset2D((DPtrs->ByAir), (DPtrs->ByPitch), 0, (L)*sizeof(realw), M+1), 1008);	
	
	GPU_ERROR_CHECKING("set EM field (device arrays) to zeros");
}



/* ----------------------------------------------------------------------------------------------- */
// cleanup
/* ----------------------------------------------------------------------------------------------- */

void Device_cleanup(DArrays* DPtrs, RxPos *d_xyzRx, int **RxLst, ModelGrid *d_MG, GridConv *d_GC) 
{

	TRACE("Cleanup_device");

  // frees memory on GPU
  print_CUDA_error_if_any( hipFree( (DPtrs->Ex) ), 3101);
	print_CUDA_error_if_any( hipFree( (DPtrs->Ey) ), 3102);
	print_CUDA_error_if_any( hipFree( (DPtrs->Ez) ), 3103);
	print_CUDA_error_if_any( hipFree( (DPtrs->Bx) ), 3104);
	print_CUDA_error_if_any( hipFree( (DPtrs->By) ), 3105);
	print_CUDA_error_if_any( hipFree( (DPtrs->Bz) ), 3106);
	print_CUDA_error_if_any( hipFree( DPtrs->BxAir ), 3107);
	print_CUDA_error_if_any( hipFree( DPtrs->ByAir ), 3108);
	
	print_CUDA_error_if_any( hipFree( (DPtrs->Con) ), 3109);
	
	print_CUDA_error_if_any( hipFree( *RxLst ), 3110);
	
	print_CUDA_error_if_any( hipFree( d_xyzRx->x ), 3111);
	print_CUDA_error_if_any( hipFree( d_xyzRx->y ), 3112);
	print_CUDA_error_if_any( hipFree( d_xyzRx->z ), 3113);
	
	print_CUDA_error_if_any( hipFree( d_xyzRx->ix ), 3114);
	print_CUDA_error_if_any( hipFree( d_xyzRx->iy ), 3115);
	print_CUDA_error_if_any( hipFree( d_xyzRx->iz ), 3116);
	
	print_CUDA_error_if_any( hipFree( d_MG->dx ), 3117);
	print_CUDA_error_if_any( hipFree( d_MG->dy ), 3118);
	print_CUDA_error_if_any( hipFree( d_MG->dz ), 3119);
	
	print_CUDA_error_if_any( hipFree( d_MG->X_Bzold ), 3120);
	print_CUDA_error_if_any( hipFree( d_MG->Y_Bzold ), 3121);
	print_CUDA_error_if_any( hipFree( d_MG->Z ), 3122);
	print_CUDA_error_if_any( hipFree( d_MG->X_BzNew ), 3123);
	print_CUDA_error_if_any( hipFree( d_MG->Y_BzNew ), 3124);
	
	print_CUDA_error_if_any( hipFree( d_GC->xBzNew2old ), 3125);
	print_CUDA_error_if_any( hipFree( d_GC->yBzNew2old ), 3126);
	print_CUDA_error_if_any( hipFree( d_GC->xBzold2New ), 3127);
	print_CUDA_error_if_any( hipFree( d_GC->yBzold2New ), 3128);
	print_CUDA_error_if_any( hipFree( d_GC->xoldBx2NewBz ), 3129);
	print_CUDA_error_if_any( hipFree( d_GC->yoldBy2NewBz ), 3130);
	
	GPU_ERROR_CHECKING("Clean arrays on device");	
}

//cpy data to host for fftw
#ifdef FFTW
void CP2Host_bz0_UPP(DArrays* DPtrs, realw *bz0, int M, int L)
{
	TRACE("cpy top slice of Bz to host to do upward continuation");
	
	print_CUDA_error_if_any( hipMemcpy( bz0, DPtrs->Bz, L*M*sizeof(realw), hipMemcpyDeviceToHost), 1201);
	
	GPU_ERROR_CHECKING("cpy top slice to host");
#if MAXDEBUG == 1
	for(int j=0; j<M; j++)
	{
		for(int i=0; i<L; i++)
		{
			if( isnan(bz0[j*M+i]) )
			{
				printf("Error in bz0 j=%d,i=%d\n",j,i);
				pause();
			}
		}
	}
#endif	
}

void CP2Device_bxby_UPP(DArrays* DPtrs, realw *bx0, realw *by0, int M, int L)
{
	TRACE("cpy BxAir and ByAir to device");
#if MAXDEBUG == 1
	for(int j=0; j<M; j++)
	{
		for(int i=0; i<L; i++)
		{
			if( isnan(bx0[j*M+i]) || isnan(by0[j*(M+1)+i]) )
			{
				printf("Error in BxAir or ByAir j=%d,i=%d\n",j,i);
				pause();
			}
		}
	}
#endif	
	print_CUDA_error_if_any( hipMemcpy2D( DPtrs->BxAir, (DPtrs->BxPitch), bx0, (L+1)*sizeof(realw), 
													 (L+1)*sizeof(realw), M, hipMemcpyHostToDevice), 1202);
	
	print_CUDA_error_if_any( hipMemcpy2D( DPtrs->ByAir, (DPtrs->ByPitch), by0, (L)*sizeof(realw), 
													 (L)*sizeof(realw), M+1, hipMemcpyHostToDevice), 1203);		
													 										 
	GPU_ERROR_CHECKING("cpy BxAir ByAir to device");												 
}
#endif

